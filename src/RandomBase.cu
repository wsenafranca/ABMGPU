#include "RandomBase.cuh"

void RandomBase::generate(unsigned int *random, const unsigned int size) {
	hiprandGenerate(p_gen, random, size);
}

void RandomBase::generateNormal(float *random, const unsigned int size, float mean=0, float stdev=1) {
	hiprandGenerateNormal(p_gen, random, size, mean, stdev);
}

void RandomBase::generateUniform(float *random, const unsigned int size) {
	hiprandGenerateUniform(p_gen, random, size);
}

void RandomBase::destroy() {
	hiprandDestroyGenerator(p_gen);
}
