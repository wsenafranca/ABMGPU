#include "RandomInt.cuh"

void RandomInt::create(long seed = 0) {
	hiprandCreateGenerator(&p_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(p_gen, seed);
}
