#include "RandomFloat.cuh"

void RandomFloat::create(long seed = 0) {
	hiprandCreateGenerator(&p_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(p_gen, seed);
}
