#include "hip/hip_runtime.h"
#include "SpatialNeighborhood.cuh"

__global__ void spatial_index_kernel(unsigned int *begins, unsigned int *ends, const unsigned int *hash, const unsigned int numAgents) {
    const int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if(tid < numAgents) {
        const int index = hash[tid];
        
        if(tid == 0 || index != hash[tid-1])
            begins[index] = tid;
        if(tid==numAgents-1 || index != hash[tid+1])
            ends[index] = tid+1;
    }
}

void SpatialNeighborhood::alloc(const unsigned int numAgents, const int2 dimension, const unsigned int neighborhood) {
	unsigned int nx = dimension.x/neighborhood;
	unsigned int ny = dimension.y/neighborhood;
	
	hipMalloc(&begins, sizeof(unsigned int)*nx*ny);
	hipMalloc(&ends, sizeof(unsigned int)*nx*ny);
	
	hipChannelFormatDesc channelDesc;
	
	channelDesc = hipCreateChannelDesc<unsigned int>();
    hipMallocArray(&beginsArray, &channelDesc, nx, ny, hipArraySurfaceLoadStore);
    hipBindTextureToArray(beginsTex, beginsArray, channelDesc);
    beginsTex.addressMode[0] = hipAddressModeBorder;
	beginsTex.addressMode[1] = hipAddressModeBorder;
    
    channelDesc = hipCreateChannelDesc<unsigned int>();
    hipMallocArray(&endsArray, &channelDesc, nx, ny, hipArraySurfaceLoadStore);
    hipBindTextureToArray(endsTex, endsArray, channelDesc);
    endsTex.addressMode[0] = hipAddressModeBorder;
	endsTex.addressMode[1] = hipAddressModeBorder;
}

void SpatialNeighborhood::free() {
	hipFree(begins);
	hipFree(ends);
	
	hipFreeArray(beginsArray);
    hipFreeArray(endsArray);
}
