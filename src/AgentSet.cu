#include "AgentSet.cuh"

#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>

void AgentSet::alloc(const unsigned int numAgents, hipStream_t stream) {
	hipMalloc(&indices, sizeof(unsigned int)*numAgents);
	hipMalloc(&alives, sizeof(bool)*numAgents);
	hipMalloc(&pregnants, sizeof(bool)*numAgents);
	
	thrust::sequence(thrust::cuda::par.on(stream), indices, indices+numAgents);
	thrust::fill(thrust::cuda::par.on(stream), alives, alives+numAgents, true);
	thrust::fill(thrust::cuda::par.on(stream), pregnants, pregnants+numAgents, false);
}

void AgentSet::resize(const unsigned int oldSize, const unsigned int newSize, hipStream_t stream) {
	
	unsigned int *h_indices;
	bool *h_alives;
	bool *h_pregnants;
	
	hipHostMalloc(&h_indices, sizeof(unsigned int)*oldSize);
	hipHostMalloc(&h_alives, sizeof(bool)*oldSize);
	hipHostMalloc(&h_pregnants, sizeof(bool)*oldSize);
	
	hipMemcpyAsync(h_indices, indices, sizeof(unsigned int)*oldSize, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_alives, alives, sizeof(bool)*oldSize, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_pregnants, pregnants, sizeof(bool)*oldSize, hipMemcpyDeviceToHost, stream);
	
	hipFree(indices);
	hipFree(alives);
	hipFree(pregnants);
	hipMalloc(&indices, sizeof(unsigned int)*newSize);
	hipMalloc(&alives, sizeof(bool)*newSize);
	hipMalloc(&pregnants, sizeof(bool)*newSize);
	
	hipMemcpyAsync(indices, h_indices, sizeof(unsigned int)*oldSize, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(alives, h_alives, sizeof(bool)*oldSize, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(pregnants, h_pregnants, sizeof(bool)*oldSize, hipMemcpyHostToDevice, stream);
	
	hipHostFree(h_indices);
	hipHostFree(h_alives);
	hipHostFree(h_pregnants);
	
	hipStreamSynchronize(stream);
}

void AgentSet::free() {
	hipFree(indices);
	hipFree(alives);
	hipFree(pregnants);
}

__device__ void AgentSet::die(const unsigned int index) {
	alives[index] = false;
}

__device__ bool AgentSet::isDead(const unsigned int index) const {
	return !alives[index];
}

__device__ const bool& AgentSet::isAlive(const unsigned int index) const {
	return alives[index];
}

__device__ void AgentSet::reproduce(const unsigned int index) {
	pregnants[index] = true;
}

__device__ const bool& AgentSet::isPregnant(const unsigned int index) const {
	return pregnants[index];
}
