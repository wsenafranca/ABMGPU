#include "AgentSet.cuh"

#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>

void AgentSet::alloc(const unsigned int numAgents, hipStream_t stream = 0) {
	hipMalloc(&indices, sizeof(unsigned int)*numAgents);
	hipMalloc(&alives, sizeof(bool)*numAgents);
	hipMalloc(&pregnants, sizeof(bool)*numAgents);
	
	thrust::sequence(thrust::cuda::par.on(stream), indices, indices+numAgents);
	thrust::fill(thrust::cuda::par.on(stream), alives, alives+numAgents, true);
	thrust::fill(thrust::cuda::par.on(stream), pregnants, pregnants+numAgents, false);
}

void AgentSet::resize(const unsigned int oldSize, const unsigned int newSize, hipStream_t stream=0) {
	
	unsigned int *h_indices;
	bool *h_alives;
	bool *h_pregnants;
	
	hipHostMalloc(&h_indices, sizeof(unsigned int)*oldSize);
	hipHostMalloc(&h_alives, sizeof(bool)*oldSize);
	hipHostMalloc(&h_pregnants, sizeof(bool)*oldSize);
	
	hipMemcpyAsync(h_indices, indices, sizeof(unsigned int)*oldSize, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_alives, alives, sizeof(bool)*oldSize, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_pregnants, pregnants, sizeof(bool)*oldSize, hipMemcpyDeviceToHost, stream);
	
	hipFree(indices);
	hipFree(alives);
	hipFree(pregnants);
	hipMalloc(&indices, sizeof(unsigned int)*newSize);
	hipMalloc(&alives, sizeof(bool)*newSize);
	hipMalloc(&pregnants, sizeof(bool)*newSize);
	
	hipMemcpyAsync(indices, h_indices, sizeof(unsigned int)*oldSize, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(alives, h_alives, sizeof(bool)*oldSize, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(pregnants, h_pregnants, sizeof(bool)*oldSize, hipMemcpyHostToDevice, stream);
	
	hipHostFree(h_indices);
	hipHostFree(h_alives);
	hipHostFree(h_pregnants);
	
	hipStreamSynchronize(stream);
}

void AgentSet::free() {
	hipFree(indices);
	hipFree(alives);
	hipFree(pregnants);
}

__device__ void AgentSet::copy(const unsigned int index1, const AgentSet *in, const unsigned int index2) {
	indices[index1] = index1;
	alives[index1] = in->alives[index2];
	pregnants[index1] = in->pregnants[index2];
}

__device__ void AgentSet::rebirth(const unsigned int index, const unsigned int parent) {
	indices[index] = index;
	alives[index] = true;
	pregnants[index] = false;
}

__device__ void AgentSet::die(const unsigned int index) {
	alives[index] = false;
}

__device__ bool AgentSet::isDead(const unsigned int index) const {
	return !alives[index];
}

__device__ const bool& AgentSet::isAlive(const unsigned int index) const {
	return alives[index];
}

__device__ void AgentSet::reproduce(const unsigned int index) {
	pregnants[index] = true;
}

__device__ const bool& AgentSet::isPregnant(const unsigned int index) const {
	return pregnants[index];
}
