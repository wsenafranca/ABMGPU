#include "hip/hip_runtime.h"
#include <simgpu.cuh>

//#define OBSERVER

#ifdef OBSERVER
#include <observer/image.cuh>
#define XDIM 512
#define YDIM 512
#define ITERATION 1000

#else

#define XDIM 2560
#define YDIM 2560
#define ITERATION 200

#endif

#define BOID_SIZE 2
#define NEIGHBOR 10
uint POP_SIZE;

// constants

const double cohesion = 1.0;
const double avoidance = 1.0;
const double randomness = 1.0;
const double consistency = 1.0;
const double momentum = 1.0;
const double jump = 0.7;

// utils

__device__ double _stx(double x, double width) { 
    if (x >= 0) { 
        if (x < width) return x;
        return x - width;
    } 
    return x + width;
}
  
__device__  double tdx(double x1, double x2,double width) {

    if (fabs(x1-x2) <= width / 2)
        return x1 - x2;  

    double dx = _stx(x1,width) - _stx(x2,width);
    if (dx * 2 > width) return dx - width;
    if (dx * 2 < -width) return dx + width;
    return dx;
}   

__device__  double _sty( double y, double height) { 
    if (y >= 0) { 
        if (y < height) return y; 
        return y - height; 
    }
    return y + height;
}

__device__  double tdy(double y1, double y2, double height) {

    if (fabs(y1-y2) <= height / 2)
        return y1 - y2;  // no wraparounds  -- quick and dirty check

    double dy = _sty(y1,height) - _sty(y2,height);
    if (dy * 2 > height) return dy - height;
    if (dy * 2 < -height) return dy + height;
    return dy;
}

// behaviour

class Boid : public Agent{
public:
    __device__ Boid() : dx(0), dy(0) {}
    
    double dx, dy;
    double x, y;
};

__device__ void init(Agent *ag) {
    Boid *boid = (Boid*)ag;
    boid->x = boid->cell->getX();
    boid->y = boid->cell->getY();
}

namespace Iterator {

class CellIterator{
public:
    __device__ CellIterator() {}
    
    __device__ CellIterator(const CellIterator &it) : p(it.p) {}
    
    __device__ CellIterator(uint *ptr) : p(ptr) {}
    
    __device__ CellIterator& operator++() {p++;return *this;}
    
    __device__ CellIterator operator++(int) {CellIterator tmp(*this); operator++(); return tmp;}
    
    __device__ const CellIterator& operator=(const CellIterator &it) {p = it.p; return *this;}
    
    __device__ bool operator==(const CellIterator& rhs) {return p==rhs.p;}
    
    __device__ bool operator!=(const CellIterator& rhs) {return p!=rhs.p;}
    
    __device__ uint& operator*() {return *p;}
    
private:
    uint *p;
};
}

namespace Collection{

class Neighborhood{
public:
    __device__ Neighborhood(Agent *ag, uint *neighborhood, uint n, uint m, uint nxdim, uint nydim) {
        uint2 pos = ag->cell->getPos();
        int x = truncf(pos.x/m);
        int y = truncf(pos.y/n);
        neighs = 0;
        for(int ny = y-1; ny <= y+1; ny++) {
            for(int nx = x-1; nx <= x+1; nx++) {
                if(nx >= 0 && nx < nxdim && ny >= 0 && ny < nydim) {
                    uint begin = tex2D(beginsRef, nx, ny);
                    uint end = tex2D(endsRef, nx, ny);
                    if(end > 0) {
                        begins[neighs] = Iterator::CellIterator(&neighborhood[begin]);
                        ends[neighs] = Iterator::CellIterator(&neighborhood[end]);
                        neighs++;
                    }
                }
            }
        }
    }
    __device__ ~Neighborhood() {
    }
    __device__ const Iterator::CellIterator& begin(uint idx) {
        return begins[idx];
    }
    __device__ const Iterator::CellIterator& end(uint idx) {
        return ends[idx];
    }
    Iterator::CellIterator begins[9];
    Iterator::CellIterator ends[9];
    uint neighs;
};

}

#define forEachNeighborhood(ag, nb) \
    Collection::Neighborhood nb(ag, neighborhood, n, m, nxdim, nydim);\
    Iterator::CellIterator it; \
    uint neighId; \
    for(uint i = 0; i < nb.neighs; i++)  \
        for(it = nb.begin(i), neighId = ((&past[*(it)] == ag) ? *(++it) : (*it)); it != nb.end(i); \
                              neighId = ((&past[*(++it)] == ag) ? *(++it) : (*it))) \
    
template<class A, class C>
__global__ void flockers(A *agents, A *past, uint size, C* cells, uint *neighborhood, uint n, uint m, uint nxdim, uint nydim) {
    uint idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx < size) {
        A *ag = &agents[idx];
        //value for collect info
		int num_neighbor = 0;
		int num_non_dead = 0;
		double cons_x = 0.0;
		double cons_y = 0.0;
		double cohe_x = 0.0;
		double cohe_y = 0.0;
		double avoid_x = 0.0;
		double avoid_y = 0.0;
		
		double	me_x = ag->x;
        double	me_y = ag->y;
        double 	old_dx = ag->dx;
        double  old_dy = ag->dy;
        
        double	him_x = 0.0;
        double	him_y = 0.0;
        double  his_dx = 0.0;
        double  his_dy = 0.0;
        int isdead = 0;
		
        double temp_tdx = XDIM;
        double temp_tdy = YDIM;
        double len = 0.0;

        Collection::Neighborhood nb(ag, neighborhood, n, m, nxdim, nydim);
        Iterator::CellIterator it;
        
        for(uint i = 0; i < nb.neighs; i++) {
            for(it = nb.begin(i); it != nb.end(i); ++it) {
                A *ag2 = &past[*it];
                if(ag == ag2) continue;
                
                him_x = ag2->x;
                him_y = ag2->y;
                his_dx = ag2->dx;
                his_dy = ag2->dy;

                temp_tdx =  tdx(me_x,him_x,XDIM);
                temp_tdy =  tdy(me_y,him_y,YDIM);

                len = hypot(temp_tdx,temp_tdy);
                if(len <= NEIGHBOR)
                {
                    double temp_value =  (powf(len,4) + 1);
	                double temp_avoidance_x = temp_tdx/temp_value;
	                double temp_avoidance_y = temp_tdy/temp_value;

	                if(isdead==0)
	                {
		                cons_x += his_dx;
		                cons_y += his_dy;
		                cohe_x += temp_tdx;
		                cohe_y += temp_tdy;
		                num_non_dead ++;
	                }

	                avoid_x += temp_avoidance_x;
	                avoid_y += temp_avoidance_y;
	                num_neighbor++;
                }
            }
        }
        
        double rand_x = fma (cuRandom(idx),2.0,-1.0);//random[index*2]*2-1.0;
		double rand_y = fma (cuRandom(idx),2.0,-1.0);
		
		double rand_length = hypot(rand_x,rand_y);
		rand_x = 0.05*rand_x/rand_length;
		rand_y = 0.05*rand_y/rand_length;
		
		if (num_non_dead > 0)
		{ 
			cohe_x = cohe_x/num_non_dead; 
			cohe_y = cohe_y/num_non_dead; 

			cons_x = cons_x/num_non_dead; 
			cons_y = cons_y/num_non_dead; 			
		}

		if(num_neighbor > 0)
		{
			avoid_x = avoid_x/num_neighbor;
			avoid_y = avoid_y/num_neighbor;
		}

		cohe_x = -cohe_x/10;
		cohe_y = -cohe_y/10;	
		avoid_x = 400*avoid_x;
		avoid_y = 400*avoid_y;

		double my_dx = cohesion * cohe_x + avoidance * avoid_x + consistency* cons_x + randomness * rand_x + momentum *old_dx; 
		double my_dy = cohesion * cohe_y + avoidance * avoid_y + consistency* cons_y + randomness * rand_y + momentum *old_dy;
           
		double dis = hypot(my_dx,my_dy);

		if (dis>0)
		{
				double value = jump / dis;
				my_dx = my_dx *value;
				my_dy = my_dy *value;
		}
		
		double rx = _stx(me_x + my_dx, XDIM);
	    double ry = _sty(me_y + my_dy, YDIM);
        int cx = (int)roundf(rx);
        int cy = (int)roundf(ry);
        if(cx >= 0 && cx < XDIM && cy >= 0 && cy < YDIM) {
            int newcid = cy*XDIM + cx;
            ag->move(&cells[newcid]);
        }
        ag->x = rx;
        ag->y = ry;
        ag->dx = my_dx;
        ag->dy = my_dy;
    }
}

#ifdef OBSERVER

template<class A>
__global__ void count(A *agents, uint size, uint *map) {
    uint i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < size) {
        map[agents[i].cell->cid] = 1;
    }
}

template<class A>
void draw(uint i, Society<A> *soc, uint *map) {
    uint *d_map;
    hipMalloc(&d_map, sizeof(uint)*XDIM*YDIM);
    hipMemset(d_map, 0, sizeof(uint)*XDIM*YDIM);
    
    uint blocks = BLOCKS(soc->size);
    count<<<blocks, THREADS>>>(soc->getAgentsDevice(), soc->size, d_map);
    
    hipMemcpy(map, d_map, sizeof(uint)*XDIM*YDIM, hipMemcpyDeviceToHost);
    
    static char filename[128];
    sprintf(filename, "img%d.bmp", i);
    saveBMP(map, XDIM, YDIM, filename);
    
    hipFree(d_map);
}

#endif

void run() {
    Random::randomObj = new Random(1234, POP_SIZE);
    
    Society<Boid> soc(POP_SIZE, POP_SIZE);
    CellularSpace<Cell> cs(XDIM, YDIM);    
    Neighborhood<Boid, Cell> nb(&soc, &cs, NEIGHBOR, NEIGHBOR);
    
    Environment::getEnvironment()->init();
    
    soc.init();
    cs.init();
    nb.init();
    
    placement(&soc, &cs);
    execute<init>(&soc);
    
#ifdef OBSERVER
    uint *map = (uint*)malloc(sizeof(uint)*XDIM*YDIM);
#endif
    
    for(int i = 1; i <= ITERATION; i++) {
        synchronize(&soc, &cs, &nb);
        flockers<<<BLOCKS(soc.size), THREADS>>>(soc.getAgentsDevice(), soc.getPastDevice(), soc.size, cs.getCellsDevice(), 
                 nb.getNeighborhoodDevice(), nb.n, nb.m, nb.neighborhoodXDim, nb.neighborhoodYDim);
#ifdef OBSERVER
        if(i % 10 == 0)
            draw(i/10, &soc, map);
#endif
    }
    
#ifdef OBSERVER
    free(map);
#endif
    
    hipDeviceSynchronize();
    Environment::getEnvironment()->reset(); // temp
}

int main() {
    hipSetDevice(1);
    
    uint pops[] = {320000, 640000, 1280000, 1600000, 2560000};
    for(int i = 0; i < 5; i++) {
        POP_SIZE = pops[i];
        
        long t = clock();
        run();
        t = clock()-t;
        printf("%lf\n", t/(double) CLOCKS_PER_SEC);
    }
    
    delete Environment::getEnvironment(); // temp
    
	return 0;
}

